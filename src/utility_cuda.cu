/* -*- mode:cuda -*-
 * LibHEOM
 * Copyright (c) Tatsushi Ikeda
 * This library is distributed under BSD 3-Clause License.
 * See LINCENSE.txt for licence.
 *------------------------------------------------------------------------*/

#include "linalg_engine/utility_cuda.h"

namespace libheom {

// The source of these messages is https://docs.nvidia.com/cuda/cublas/index.html.
std::map<hipblasStatus_t, std::string> CUBLAS_ERR_MSG = {{
  { HIPBLAS_STATUS_SUCCESS,
    "HIPBLAS_STATUS_SUCCESS: The operation completed successfully." },
  { HIPBLAS_STATUS_NOT_INITIALIZED,
    "HIPBLAS_STATUS_NOT_INITIALIZED: The cuBLAS library was not initialized. This is usually caused by the lack of a prior hipblasCreate() call, an error in the CUDA Runtime API called by the cuBLAS routine, or an error in the hardware setup." },
  { HIPBLAS_STATUS_ALLOC_FAILED,
    "HIPBLAS_STATUS_ALLOC_FAILED: Resource allocation failed inside the cuBLAS library. This is usually caused by a hipMalloc() failure." },
  { HIPBLAS_STATUS_INVALID_VALUE,
    "HIPBLAS_STATUS_INVALID_VALUE: An unsupported value or parameter was passed to the function (a negative vector size, for example)." },
  { HIPBLAS_STATUS_ARCH_MISMATCH,
    "HIPBLAS_STATUS_ARCH_MISMATCH: The function requires a feature absent from the device architecture; usually caused by the lack of support for double precision." },
  { HIPBLAS_STATUS_MAPPING_ERROR,
    "HIPBLAS_STATUS_MAPPING_ERROR: An access to GPU memory space failed, which is usually caused by a failure to bind a texture." },
  { HIPBLAS_STATUS_EXECUTION_FAILED,
    "HIPBLAS_STATUS_EXECUTION_FAILED: The GPU program failed to execute. This is often caused by a launch failure of the kernel on the GPU, which can be caused by multiple reasons." },
  { HIPBLAS_STATUS_INTERNAL_ERROR,
    "HIPBLAS_STATUS_INTERNAL_ERROR: An internal cuBLAS operation failed. This error is usually caused by a hipMemcpyAsync() failure." },
  { HIPBLAS_STATUS_NOT_SUPPORTED,
    "HIPBLAS_STATUS_NOT_SUPPORTED: The functionnality requested is not supported." },
  { HIPBLAS_STATUS_UNKNOWN,
    "HIPBLAS_STATUS_UNKNOWN: The functionnality requested requires some license and an error was detected when trying to check the current licensing. This error can happen if the license is not present or is expired or if the environment variable NVIDIA_LICENSE_FILE is not set properly." }}};


// The source of these messages is from https://docs.nvidia.com/cuda/cusparse/index.html.
std::map<hipsparseStatus_t, std::string> CUSPARSE_ERR_MSG = {{
  { HIPSPARSE_STATUS_SUCCESS,
    "HIPSPARSE_STATUS_SUCCESS: The operation completed successfully." },
  { HIPSPARSE_STATUS_NOT_INITIALIZED,
    "HIPSPARSE_STATUS_NOT_INITIALIZED: The cuSPARSE library was not initialized. This is usually caused by the lack of a prior call, an error in the CUDA Runtime API called by the cuSPARSE routine, or an error in the hardware setup." },
  { HIPSPARSE_STATUS_ALLOC_FAILED,
    "HIPSPARSE_STATUS_ALLOC_FAILED: Resource allocation failed inside the cuSPARSE library. This is usually caused by a hipMalloc() failure." },
  { HIPSPARSE_STATUS_INVALID_VALUE,
    "HIPSPARSE_STATUS_INVALID_VALUE: An unsupported value or parameter was passed to the function (a negative vector size, for example)." },
  { HIPSPARSE_STATUS_ARCH_MISMATCH,
    "HIPSPARSE_STATUS_ARCH_MISMATCH: The function requires a feature absent from the device architecture; usually caused by the lack of support for atomic operations or double precision." },
  { HIPSPARSE_STATUS_MAPPING_ERROR,
    "HIPSPARSE_STATUS_MAPPING_ERROR: An access to GPU memory space failed, which is usually caused by a failure to bind a texture." },
  { HIPSPARSE_STATUS_EXECUTION_FAILED,
    "HIPSPARSE_STATUS_EXECUTION_FAILED: The GPU program failed to execute. This is often caused by a launch failure of the kernel on the GPU, which can be caused by multiple reasons." },
  { HIPSPARSE_STATUS_INTERNAL_ERROR,
    "HIPSPARSE_STATUS_INTERNAL_ERROR: An internal cuSPARSE operation failed. This error is usually caused by a hipMemcpyAsync() failure." },
  { HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED,
    "HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED: The matrix type is not supported by this function. This is usually caused by passing an invalid matrix descriptor to the function." }}};


// The source of these messages is https://docs.nvidia.com/cuda/cusolver/index.html
std::map<hipsolverStatus_t, std::string> CUSOLVER_ERR_MSG = {{
  { HIPSOLVER_STATUS_SUCCESS,
    "HIPSOLVER_STATUS_SUCCESS: The operation completed successfully." },
  { HIPSOLVER_STATUS_NOT_INITIALIZED,
    "HIPSOLVER_STATUS_NOT_INITIALIZED: The cuSolver library was not initialized. This is usually caused by the lack of a prior call, an error in the CUDA Runtime API called by the cuSolver routine, or an error in the hardware setup." },
  { HIPSOLVER_STATUS_ALLOC_FAILED,
    "HIPSOLVER_STATUS_ALLOC_FAILED: Resource allocation failed inside the cuSolver library. This is usually caused by a hipMalloc() failure." },
  { HIPSOLVER_STATUS_INVALID_VALUE,
    "HIPSOLVER_STATUS_INVALID_VALUE: An unsupported value or parameter was passed to the function (a negative vector size, for example)." },
  { HIPSOLVER_STATUS_ARCH_MISMATCH,
    "HIPSOLVER_STATUS_ARCH_MISMATCH: The function requires a feature absent from the device architecture; usually caused by the lack of support for atomic operations or double precision." },
  { HIPSOLVER_STATUS_EXECUTION_FAILED,
    "HIPSOLVER_STATUS_EXECUTION_FAILED: The GPU program failed to execute. This is often caused by a launch failure of the kernel on the GPU, which can be caused by multiple reasons." },
  { HIPSOLVER_STATUS_INTERNAL_ERROR,
    "HIPSOLVER_STATUS_INTERNAL_ERROR: An internal cuSolver operation failed. This error is usually caused by a hipMemcpyAsync() failure." },
  { HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED,
    "HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED: The matrix type is not supported by this function. This is usually caused by passing an invalid matrix descriptor to the function." },
}};

} // namespace libheom
