/*
 * LibHEOM: Copyright (c) Tatsushi Ikeda
 * This library is distributed under BSD 3-Clause License.
 * See LINCENSE.txt for licence.
 *------------------------------------------------------------------------*/

#include "handle_gpu.h"

#include "gpu_info.h"

namespace libheom {

void handle_gpu::init(int device_number) {
  this->device_number = device_number;
  SetGpuDevice(device_number);
  
  CUSPARSE_CALL(hipsparseCreate(&cusparse));
  CUBLAS_CALL(hipblasCreate(&cublas));
  
  CUSPARSE_CALL(hipsparseCreateMatDescr(&mat_descr));
  CUSPARSE_CALL(hipsparseSetMatType(mat_descr, HIPSPARSE_MATRIX_TYPE_GENERAL));
  CUSPARSE_CALL(hipsparseSetMatIndexBase(mat_descr, HIPSPARSE_INDEX_BASE_ZERO));
}

}