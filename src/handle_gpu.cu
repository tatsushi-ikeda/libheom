/*
 * LibHEOM, version 0.5
 * Copyright (c) 2019-2020 Tatsushi Ikeda
 *
 * This library is distributed under BSD 3-Clause License.
 * See LINCENSE.txt for licence.
 *------------------------------------------------------------------------*/

#include "handle_gpu.h"

#include "gpu_info.h"

namespace libheom {

void HandleGpu::Initialize(int device_number) {
  this->device_number = device_number;
  SetGpuDevice(device_number);
  
  CUSPARSE_CALL(hipsparseCreate(&cusparse));
  CUBLAS_CALL(hipblasCreate(&cublas));
  
  CUSPARSE_CALL(hipsparseCreateMatDescr(&mat_descr));
  CUSPARSE_CALL(hipsparseSetMatType(mat_descr, HIPSPARSE_MATRIX_TYPE_GENERAL));
  CUSPARSE_CALL(hipsparseSetMatIndexBase(mat_descr, HIPSPARSE_INDEX_BASE_ZERO));
}

}