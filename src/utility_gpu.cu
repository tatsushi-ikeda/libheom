/*
 * LibHEOM: Copyright (c) Tatsushi Ikeda
 * This library is distributed under BSD 3-Clause License.
 * See LINCENSE.txt for licence.
 *------------------------------------------------------------------------*/

#include "utility_gpu.h"

namespace libheom {

// The source of these messages is https://docs.nvidia.com/cuda/cublas/index.html.
std::map<hipblasStatus_t,std::string> CUBLAS_ERR_MSG =
  {{
      {HIPBLAS_STATUS_SUCCESS,
       "HIPBLAS_STATUS_SUCCESS: The operation completed successfully."},
      {HIPBLAS_STATUS_NOT_INITIALIZED,
       "HIPBLAS_STATUS_NOT_INITIALIZED: The cuBLAS library was not initialized. This is usually caused by the lack of a prior hipblasCreate() call, an error in the CUDA Runtime API called by the cuBLAS routine, or an error in the hardware setup."},
      {HIPBLAS_STATUS_ALLOC_FAILED,
       "HIPBLAS_STATUS_ALLOC_FAILED: Resource allocation failed inside the cuBLAS library. This is usually caused by a hipMalloc() failure."},
      {HIPBLAS_STATUS_INVALID_VALUE,
       "HIPBLAS_STATUS_INVALID_VALUE: An unsupported value or parameter was passed to the function (a negative vector size, for example)."},
      {HIPBLAS_STATUS_ARCH_MISMATCH,
       "HIPBLAS_STATUS_ARCH_MISMATCH: The function requires a feature absent from the device architecture; usually caused by the lack of support for double precision."},
      {HIPBLAS_STATUS_MAPPING_ERROR,
       "HIPBLAS_STATUS_MAPPING_ERROR: An access to GPU memory space failed, which is usually caused by a failure to bind a texture."},
      {HIPBLAS_STATUS_EXECUTION_FAILED,
       "HIPBLAS_STATUS_EXECUTION_FAILED: The GPU program failed to execute. This is often caused by a launch failure of the kernel on the GPU, which can be caused by multiple reasons."},
      {HIPBLAS_STATUS_INTERNAL_ERROR,
       "HIPBLAS_STATUS_INTERNAL_ERROR: An internal cuBLAS operation failed. This error is usually caused by a hipMemcpyAsync() failure."},
      {HIPBLAS_STATUS_NOT_SUPPORTED,
       "HIPBLAS_STATUS_NOT_SUPPORTED: The functionnality requested is not supported."},
      {HIPBLAS_STATUS_UNKNOWN,
       "HIPBLAS_STATUS_UNKNOWN: The functionnality requested requires some license and an error was detected when trying to check the current licensing. This error can happen if the license is not present or is expired or if the environment variable NVIDIA_LICENSE_FILE is not set properly."}
    }};


// The source of these messages is from https://docs.nvidia.com/cuda/cusparse/index.html.
std::map<hipsparseStatus_t,std::string> CUSPARSE_ERR_MSG =
  {{
      {HIPSPARSE_STATUS_SUCCESS,
       "HIPSPARSE_STATUS_SUCCESS: The operation completed successfully."},
      {HIPSPARSE_STATUS_NOT_INITIALIZED,
       "HIPSPARSE_STATUS_NOT_INITIALIZED: The cuSPARSE library was not initialized. This is usually caused by the lack of a prior call, an error in the CUDA Runtime API called by the cuSPARSE routine, or an error in the hardware setup."},
      {HIPSPARSE_STATUS_ALLOC_FAILED,
       "HIPSPARSE_STATUS_ALLOC_FAILED: Resource allocation failed inside the cuSPARSE library. This is usually caused by a hipMalloc() failure."},
      {HIPSPARSE_STATUS_INVALID_VALUE,
       "HIPSPARSE_STATUS_INVALID_VALUE: An unsupported value or parameter was passed to the function (a negative vector size, for example)."},
      {HIPSPARSE_STATUS_ARCH_MISMATCH,
       "HIPSPARSE_STATUS_ARCH_MISMATCH: The function requires a feature absent from the device architecture; usually caused by the lack of support for atomic operations or double precision."},
      {HIPSPARSE_STATUS_MAPPING_ERROR,
       "HIPSPARSE_STATUS_MAPPING_ERROR: An access to GPU memory space failed, which is usually caused by a failure to bind a texture."},
      {HIPSPARSE_STATUS_EXECUTION_FAILED,
       "HIPSPARSE_STATUS_EXECUTION_FAILED: The GPU program failed to execute. This is often caused by a launch failure of the kernel on the GPU, which can be caused by multiple reasons."},
      {HIPSPARSE_STATUS_INTERNAL_ERROR,
       "HIPSPARSE_STATUS_INTERNAL_ERROR: An internal cuSPARSE operation failed. This error is usually caused by a hipMemcpyAsync() failure."},
      {HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED,
       "HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED: The matrix type is not supported by this function. This is usually caused by passing an invalid matrix descriptor to the function."}
    }};

}